#include "hipblas.h"

hipblasHandle_t handle;
hipblasCreate(&handle);

// Déclaration des matrices host et device
float *h_A, *h_B, *h_C;
float *d_A, *d_B, *d_C;
int m, n, k; // dimensions des matrices

// Allocation mémoire sur le host et le device, et initialisation des données

// Copie des données du host vers le device
hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

// Exécution de la multiplication de matrices sur le device avec cuBLAS
const float alpha = 1.0f;
const float beta = 0.0f;
hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);

// Copie des résultats du device vers le host
hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

hipblasDestroy(handle);