#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main() {
    // Initialiser cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Définir la taille des matrices
    int m = 3;
    int n = 3;
    int k = 3;

    // Allouer et initialiser les matrices sur le CPU
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(m * k * sizeof(float));
    h_B = (float*)malloc(k * n * sizeof(float));
    h_C = (float*)malloc(m * n * sizeof(float));

    // Initialiser les matrices avec des valeurs quelconques

    // Allouer de la mémoire sur le GPU
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    // Copier les données du CPU vers le GPU
    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);

    // Effectuer la multiplication de matrices sur le GPU avec cuBLAS
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);

    // Copier le résultat du GPU vers le CPU
    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Afficher le résultat
    printf("Résultat de la multiplication de matrices :\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%f ", h_C[i * n + j]);
        }
        printf("\n");
    }

    // Libérer la mémoire
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Détruire le gestionnaire cuBLAS
    hipblasDestroy(handle);

    return 0;
}
